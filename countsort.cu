
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#define LISTSIZE 10000000
#define MAXNUM 10000
#define THREAD_PER_BLOCK 1024

__global__ void gpu_countsort(int* globalTable_d, int* unsort){
	__shared__ int table[MAXNUM];
	if(threadIdx.x == 0) memset(table, 0, sizeof(int)*MAXNUM);
	__syncthreads();//block level synchronization
	
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if(index < LISTSIZE){
		int num = unsort[index];
		atomicAdd(&table[num-1], 1);
	}
	__syncthreads();
	if(threadIdx.x == 0){
		for(int i=0; i<MAXNUM; i++){
			atomicAdd(&(globalTable_d[i]), table[i]);
		}
	}
}

void genList(int** unsort){
	*unsort = (int*)malloc(sizeof(int) * LISTSIZE);
	for(int i=0; i<LISTSIZE; i++){
		(*unsort)[i] = rand()%MAXNUM + 1;
	}
}

int main()
{
	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int *unsort;
	genList(&unsort);

	int *unsort_d, *table_d;
	int listSize = LISTSIZE * sizeof(int);
	int tableSize = MAXNUM * sizeof(int);
	hipMalloc((void**)&unsort_d, listSize);
	hipMemcpy(unsort_d, unsort, listSize, hipMemcpyHostToDevice);
	hipMalloc((void**)&table_d, tableSize);
	hipMemset(table_d, 0, tableSize);

	int blockNum;
	blockNum = (LISTSIZE/THREAD_PER_BLOCK) + ( LISTSIZE%THREAD_PER_BLOCK==0 ?0:1 );
	hipEventRecord(start, 0);
	gpu_countsort<<<blockNum, THREAD_PER_BLOCK>>>(table_d, unsort_d);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	hipDeviceSynchronize();
	printf("time in gpu: %31.f ms\n", time);

	int *table, *sort;
	sort = (int*)malloc(listSize);
	memset(sort, 0, listSize);
	table = (int*)malloc(tableSize);
	hipMemcpy(table, table_d, tableSize, hipMemcpyDeviceToHost);

	int index=0;
	for(int i=0; i<MAXNUM; i++){
		for(int j=0; j<table[i]; j++) sort[index++] = i+1;
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf("time in cpu: %31.f ms\n", time);
//	for(int i=0; i<LISTSIZE; i++) printf("%d ", sort[i]);

	hipFree(unsort_d);
	hipFree(table_d);
	free(unsort);
	free(table);

	return 0;
}
